
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <utility>

__global__ void add_ballot (uint64_t *T, int *B, int C) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int cutoff = blockIdx.z;
	if (B[x] > cutoff && B[y] > cutoff)
		T[cutoff *C*C + x *C + y]++;
}

__global__ void calculate_scores (double *S, uint64_t *T, int C, int A_length, int *A) {
	int C1 = blockIdx.x;
	int C2 = blockIdx.y;
	int cutoff = blockIdx.z;
	S[cutoff *C*C + C1 *C + C2] = 0.0;
	uint64_t V1 = T[cutoff *C*C + C1 *C + C1];
	uint64_t V2 = T[cutoff *C*C + C2 *C + C2];
	if (V1 == 0 || V2 == 0)
		return;
	double D1 = 1.0;
	double D2 = 1.0;
	for (int i = 0; i < A_length; i++) {
		uint64_t VA = T[cutoff *C*C + A[i] *C + A[i]];
		if (VA == 0)
			return;
		D1 += 1.0 * T[cutoff *C*C + C1 *C + A[i]] / VA;
		D2 += 1.0 * T[cutoff *C*C + C2 *C + A[i]] / VA;
	}
	if (C1 == C2)
		S[cutoff *C*C + C1 *C + C2] = 1.0 / (D1 / V1);
	else
		S[cutoff *C*C + C1 *C + C2] = 2.0 / (D1 / V1 + D2 / V2 + 1.0 * T[cutoff *C*C + C1 *C + C2] / (V1 * V2));
}

class Tensor {
	int C;
	int M;
	uint64_t *D;
	uint64_t V;

	public:

	Tensor (int C, int M) :
		C{C},
		M{M},
		V{0}
	{
		hipMalloc((void **)&D, M * C * C * sizeof(uint64_t));
		hipMemset(D, 0, M * C * C * sizeof(uint64_t));
	}

	Tensor (const Tensor&) = delete;

	Tensor (Tensor&&) = default;

	auto vote_count () const { return V; }

	void add_ballot (int *B) {
		::add_ballot<<<1, dim3(C, C, M)>>> (D, B, C);
		V++;
	}

	int get_cutoff (double *S, double Q, int A_length, int *A) const {
		for (int cutoff = M-1; cutoff > 0; cutoff--)
			for (int i = 0; i < C; i++) {
				bool skip = false;
				for (int k = 0; k < A_length; k++)
					if (A[k] == i)
						skip = true;
				if (skip)
					continue;
				if (S[(cutoff-1) *C*C + i *C + i] >= Q)
					return cutoff;
			}
		return 0;
	}

	int get_cutoff (double *S, double Q, int A_length, int *A, bool (*are_valid)(int, int)) const {
		for (int cutoff = M-1; cutoff > 0; cutoff--)
			for (int i = 0; i < C; i++)
				for (int j = 0; j < i; j++) {
					if (!are_valid(i, j))
						continue;
					bool skip = false;
					for (int k = 0; k < A_length; k++)
						if (A[k] == i || A[k] == j)
							skip = true;
					if (skip)
						continue;
					if (S[(cutoff-1) *C*C + i *C + j] >= Q)
						return cutoff;
				}
		return 0;
	}

	int get_next (double Q, int A_length, int *A) const {
		double *_S;
		hipMalloc((void **)&_S, M * C * C * sizeof(double));
		calculate_scores<<<1,dim3(C, C, M)>>> (_S, D, C, A_length, A);
		double *S = new double[M * C * C];
		hipMemcpy(S, &_S, M * C * C * sizeof(double), hipMemcpyDeviceToHost);
		hipFree(_S);
		auto idx = new int[C];
		int idx_length = 0;
		for (int cutoff = get_cutoff(S, Q, A_length, A); cutoff < M; cutoff++) {
			if (idx_length == 0)
				for (int i = 0; i < C; i++) {
					bool skip = false;
					for (int k = 0; k < A_length; k++)
						if (A[k] == i)
							skip = true;
					if (skip)
						continue;
					if (cutoff == 0 || S[(cutoff-1) *C*C + i *C + i] >= Q)
						idx[idx_length++] = i;
				}
			else if (idx_length == 1)
				break;
			double val = -1.0;
			auto idx2 = new int[idx_length];
			int idx2_length = 0;
			for (int i = 0; i < idx_length; i++) {
				double cur = S[cutoff *C*C + idx[i] *C + idx[i]];
				if (cur == val)
					idx2[idx2_length++] = idx[i];
				else if (cur > val) {
					val = cur;
					idx2_length = 1;
					idx2[0] = idx[i];
				}
			}
			delete[] idx;
			idx = idx2;
		}
		auto r = idx[0];
		delete[] idx;
		delete[] S;
		return r;
	}

	std::pair<int, int> get_next (double Q, int A_length, int *A, bool (*are_valid)(int, int)) const {
		int *_A;
		double *_S;
		hipMalloc((void **)&_A, A_length * sizeof(int));
		hipMemcpy(_A, A, A_length * sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void **)&_S, M * C * C * sizeof(double));
		calculate_scores<<<1,dim3(C, C, M)>>> (_S, D, C, A_length, _A);
		double *S = new double[M * C * C];
		hipMemcpy(S, _S, M * C * C * sizeof(double), hipMemcpyDeviceToHost);
		hipFree(_S);
		hipFree(_A);
		auto idx = new std::pair<int, int>[C];
		int idx_length = 0;
		for (int cutoff = get_cutoff(S, Q, A_length, A, are_valid); cutoff < M; cutoff++) {
			if (idx_length == 0)
				for (int i = 0; i < C; i++)
					for (int j = 0; j < i; j++) {
						if (!are_valid(i, j))
							continue;
						bool skip = false;
						for (int k = 0; k < A_length; k++)
							if (A[k] == i || A[k] == j)
								skip = true;
						if (skip)
							continue;
						if (cutoff == 0 || S[(cutoff-1) *C*C + i *C + j] >= Q)
							idx[idx_length++] = {i, j};
					}
			else if (idx_length == 1)
				break;
			double val = -1.0;
			auto idx2 = new std::pair<int, int>[idx_length];
			int idx2_length = 0;
			for (int i = 0; i < idx_length; i++) {
				double cur = S[cutoff *C*C + idx[i].first *C + idx[i].second];
				if (cur == val)
					idx2[idx2_length++] = idx[i];
				else if (cur > val) {
					val = cur;
					idx2_length = 1;
					idx2[0] = idx[i];
				}
			}
			delete[] idx;
			idx = idx2;
		}
		auto r = idx[0];
		delete[] idx;
		delete[] S;
		return r;
	}

	int *get_winners (int W, bool (*are_valid)(int, int)) const {
		auto r = new int[W];
		int l = 0;
		while (l < W-1) {
			auto n = get_next(1.0 * vote_count () / W, l, r, are_valid);
			r[l++] = n.first;
			r[l++] = n.second;
		}
		while (l < W-1)
			r[l++] = get_next(1.0 * vote_count () / W, l, r);
		return r;
	}
};
